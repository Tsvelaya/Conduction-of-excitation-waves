//                                                                   -*- C++ -*-
#include <cassert>
#include <cstdio>

#include "util.h"
#include "debug.h"
#include "hip/hip_runtime.h"

extern "C"
void *xmalloc(size_t size, location_t location)
{
    void *p;
    hipError_t e;

    switch (location) {
    case HOST:
        p = malloc(size);
        assert(p);
        return p;
    case DEVICE:
        e = hipMalloc(&p, size);
        check(e);
        e = hipMemset(p, 0, size);
        check(e);
        return p;
    case LOCKED:
	e = hipHostMalloc(&p, size);
	check(e);
	return p;
    default:
        assert(0);
    }

    return NULL;
}

extern "C"
void xfree(void *p, location_t location)
{
    switch (location) {
    case HOST:
        free(p);
        return;
    case DEVICE:
        hipFree(p);
        return;
    default:
        assert(0);
    }
}

static hipMemcpyKind kind_of_dir(direction_t dir)
{
    switch (dir) {
    case HOST_TO_HOST: return hipMemcpyHostToHost;
    case HOST_TO_DEVICE: return hipMemcpyHostToDevice;
    case DEVICE_TO_HOST: return hipMemcpyDeviceToHost;
    case DEVICE_TO_DEVICE: return hipMemcpyDeviceToDevice;
    default: assert(0);
    }

    return hipMemcpyDefault; 			// Never get here
}

extern "C"
void xmemcpy(void *dst, void *src, size_t n, direction_t dir)
{
    hipError_t e;
    if (dir == DEVICE_TO_HOST_ASYNC) {
	e = hipMemcpyAsync(dst, src, n, hipMemcpyDeviceToHost,
			    Cuda_Stream[COPYING]);
	check(e);
	e = hipStreamSynchronize(Cuda_Stream[COPYING]);
	check(e);
    } else {
	e = hipMemcpy(dst, src, n, kind_of_dir(dir));
	check(e);
    }
}
