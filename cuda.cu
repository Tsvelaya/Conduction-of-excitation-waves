//                                                                   -*- C++ -*-
#include "config.h"
#include "debug.h"
#include "util.h"

hipStream_t Cuda_Stream[2];

extern int device_num;

extern "C"
void init_cuda(void)
{
    int i;
    hipError_t e;
    int dev;

    hipGetDeviceCount(&dev);
    if (device_num < dev)
	hipSetDevice(device_num);

#if PRECISION == FLOAT
    e = hipDeviceSetSharedMemConfig(hipSharedMemBankSizeFourByte);
#elif PRECISION == DOUBLE
    e = hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
#endif
    check(e);

    for (i = 0; i < SIZE(Cuda_Stream); i++)
	hipStreamCreate(&Cuda_Stream[i]);
}
